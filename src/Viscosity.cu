#include "hip/hip_runtime.h"
#include "Main.cuh"

extern int NSEC, size_grid, NRAD;

extern float TRANSITIONWIDTH, TRANSITIONRADIUS, TRANSITIONRATIO, ASPECTRATIO, LAMBDADOUBLING;
extern float VISCOSITY, CAVITYRATIO, CAVITYRADIUS, CAVITYWIDTH, ALPHAVISCOSITY;
extern float ViscosityAlpha, onethird, invdphi;

extern float *SoundSpeed, *GLOBAL_bufarray, *viscosity_array, *invRmed, *Rmed, *Rsup, *VradInt, *VthetaInt;
extern float *SoundSpeed_d, *viscosity_array_d, *invdiffRmed_d, *Rinf_d, *invRinf_d, *invRmed_d, *Rmed_d;
extern float *invdiffRsup_d, *Rsup_d, *Dens_d, *Vrad_d, *Vtheta_d, *Vradial_d, *Vazimutal_d, *VthetaInt_d, *VradInt_d;

float *DivergenceVelocity, *DRP, *DRR, *DPP, *TAURR, *TAURP, *TAUPP;
float *DivergenceVelocity_d, *DRP_d, *DRR_d, *DPP_d, *TAURR_d, *TAURP_d, *TAUPP_d;
float PhysicalTime =0.0, PhysicalTimeInitial= 0.0;

extern dim3 dimGrid2, dimBlock2;

__host__ float FViscosity(float r)
{
  float viscosity, rmin, rmax, scale;
  int i = 0;
  viscosity = VISCOSITY;
  if (ViscosityAlpha){
    while (Rmed[i] < r) i++;
    viscosity = ALPHAVISCOSITY*GLOBAL_bufarray[i] * GLOBAL_bufarray[i] * pow(r, 1.5);
  }
  rmin = CAVITYRADIUS-CAVITYWIDTH*ASPECTRATIO;
  rmax = CAVITYRADIUS+CAVITYWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) viscosity *= CAVITYRATIO;
  if ((r >= rmin) && (r <= rmax))
    viscosity *= exp((rmax-r)/(rmax-rmin)*log(CAVITYRATIO));
  return viscosity;
}

__host__ float AspectRatioHost(float r)
{
  float aspectratio, rmin, rmax, scale;
  aspectratio = ASPECTRATIO;
  rmin = TRANSITIONRADIUS-TRANSITIONWIDTH*ASPECTRATIO;
  rmax = TRANSITIONRADIUS+TRANSITIONWIDTH*ASPECTRATIO;
  scale = 1.0+(PhysicalTime-PhysicalTimeInitial)*LAMBDADOUBLING;
  rmin *= scale;
  rmax *= scale;
  if (r < rmin) aspectratio *= TRANSITIONRATIO;
  if ((r >= rmin) && (r <= rmax))
    aspectratio *= exp((rmax-r)/(rmax-rmin)*log(TRANSITIONRATIO));
  return aspectratio;
}

__host__ void InitViscosity ()
{
  DivergenceVelocity  = (float *)malloc(size_grid*sizeof(float));
  DRR                 = (float *)malloc(size_grid*sizeof(float));
  DRP                 = (float *)malloc(size_grid*sizeof(float));
  DPP                 = (float *)malloc(size_grid*sizeof(float));
  TAURR               = (float *)malloc(size_grid*sizeof(float));
  TAURP               = (float *)malloc(size_grid*sizeof(float));
  TAUPP               = (float *)malloc(size_grid*sizeof(float));
  InitViscosityDevice ();
}

__host__ void InitViscosityDevice ()
{
  gpuErrchk(hipMalloc((void**)&DivergenceVelocity_d, size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRR_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DRP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&DPP_d,                size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURR_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAURP_d,              size_grid*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&TAUPP_d,              size_grid*sizeof(float)));
}

__host__ void ComputeViscousTerms (float *Vradial_d, float *Vazimutal_d, float *Dens)
{

  if (ViscosityAlpha){
    //gpuErrchk(hipMemcpy(SoundSpeed, SoundSpeed_d, size_grid*sizeof(float), hipMemcpyDeviceToHost));
    Make1Dprofile (1);
  }

  for (int i = 0; i < NRAD; i++)
    viscosity_array[i] = FViscosity(Rmed[i]);
  gpuErrchk(hipMemcpy(viscosity_array_d, viscosity_array, (NRAD+1)*sizeof(float), hipMemcpyHostToDevice));

  ViscousTermsKernelDRP<<<dimGrid2, dimBlock2>>>(Vradial_d, Vazimutal_d , DRR_d, DPP_d, DivergenceVelocity_d,
    DRP_d, invdiffRsup_d, invRmed_d, Rsup_d, Rinf_d, invdiffRmed_d, NRAD, NSEC, invRinf_d, invdphi);
  gpuErrchk(hipDeviceSynchronize());

  ViscousTermsKernelTAURP<<<dimGrid2, dimBlock2>>>(Dens_d, viscosity_array_d, DRR_d, DPP_d, onethird, DivergenceVelocity_d,
    TAURR_d, TAUPP_d, TAURP_d, DRP_d, NRAD, NSEC);
  gpuErrchk(hipDeviceSynchronize());

}

__host__ void UpdateVelocitiesWithViscosity(float *VradInt, float *VthetaInt, float *Dens, float DeltaT)
{
  UpdateVelocitiesKernel<<<dimGrid2, dimBlock2>>>(VthetaInt_d, VradInt_d, invRmed_d, Rmed_d, Rsup_d, Rinf_d,
    invdiffRmed_d, invdiffRsup_d,  Dens_d, invRinf_d, TAURR_d, TAURP_d, TAUPP_d, DeltaT, NRAD, NSEC, invdphi);
    gpuErrchk(hipDeviceSynchronize());
}
